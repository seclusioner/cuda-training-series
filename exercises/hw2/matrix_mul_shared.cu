
#include <hip/hip_runtime.h>
#include <stdio.h>

// these are just for timing measurments
#include <time.h>
#include <cstdlib>  // rand()
#include <ctime>    // time()

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

const int DSIZE = 4096;
const int block_size = 32;  // CUDA maximum is 1024 *total* threads in block
const float EPS = 1e-5;

// matrix multiply (naive) kernel: C = A * B
// (M, N) = (M, K) @ (K, N)
__global__ void mmul(const float *A, const float *B, float *C, int M, int K, int N) {
  __shared__ float As[block_size][block_size];
  __shared__ float Bs[block_size][block_size];

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float temp = 0.0f;

  for (int i = 0; i < (K + block_size - 1) / block_size; i++) {
    int tiledCol = i * block_size + threadIdx.x;
    int tiledRow = i * block_size + threadIdx.y;

    As[threadIdx.y][threadIdx.x] = (row < M && tiledCol < K) ? A[row * K + tiledCol] : 0.0f;
    Bs[threadIdx.y][threadIdx.x] = (tiledRow < K && col < N) ? B[tiledRow * N + col] : 0.0f;

    __syncthreads();

    for (int k = 0; k < block_size; ++k) {
      temp += As[threadIdx.y][k] * Bs[k][threadIdx.x];
    }

    __syncthreads();
  }

  if (row < M && col < N) {
    C[row * N + col] = temp;
  }
}

void mmul_host(const float *A, const float *B, float *C, int m, int n, int p){
  for(int i=0;i<m;i++){
    for(int j=0;j<p;j++){
      for(int k=0;k<n;k++){
        C[i*p+j] += A[i*n+k] * B[k*p+j];
      }
    }
  }
}

int main(int argc, char* argv[]){
  if (argc != 4) {
      printf("Usage: %s <m> <n> <p>\n", argv[0]);
      return 1;
  }
  srand(time(0));

  int m = atoi(argv[1]);
  int n = atoi(argv[2]);
  int p = atoi(argv[3]);

  if (m <= 0 || n <= 0 || p <= 0 || m > DSIZE || n > DSIZE || p > DSIZE) {
      printf("Matrix sizes must be in range 1 to %d\n", DSIZE);
      return 1;
  }
  printf("%d by %d multiply %d by %d\n", m, n, n, p);

  float *h_A, *h_B, *h_C, *h_C2, *d_A, *d_B, *d_C;

  // these are just for timing
  clock_t t0, t1, t2;
  double t1sum=0.0;
  double t2sum=0.0;

  // initialize
  h_A = new float[m*n]; // m x n
  h_B = new float[n*p]; // n x p
  h_C = new float[m*p]; // m x p
  h_C2 = new float[m*p];
  for (int i = 0; i < (m*n); i++){
    h_A[i] = (float)(rand() % 10);
  }
  for (int i = 0; i < (n*p); i++){
    h_B[i] = (float)(rand() % 10);
  }
  memset(h_C, 0, sizeof(float)*m*p);
  memset(h_C2, 0, sizeof(float)*m*p);

  // start timing
  t0 = clock();
  mmul_host(h_A, h_B, h_C, m, n, p);
  t1 = clock();
  t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
  printf("CPU took %f seconds.\n", t1sum);

  // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, m*n*sizeof(float));
  hipMalloc(&d_B, n*p*sizeof(float));
  hipMalloc(&d_C, m*p*sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  hipMemcpy(d_A, h_A, m*n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, n*p*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");

  // Cuda processing sequence step 1 is complete

  // Launch kernel
  dim3 block(block_size, block_size);  // dim3 variable holds 3 dimensions
  dim3 grid((p+block.x-1)/block.x, (m+block.y-1)/block.y);
  hipMemset(d_C, 0, m * p * sizeof(float));
  mmul<<<grid, block>>>(d_A, d_B, d_C, m, n, p);
  hipDeviceSynchronize();
  cudaCheckErrors("kernel launch failure");

  // Cuda processing sequence step 2 is complete

  // Copy results back to host
  hipMemcpy(h_C2, d_C, m*p*sizeof(float), hipMemcpyDeviceToHost);

  // GPU timing
  t2 = clock();
  t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
  printf ("GPU took %f seconds\n", t2sum);

  // Cuda processing sequence step 3 is complete

  // Verify results
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  for (int i = 0; i < m*p; i++) {
    if (fabs(h_C[i] - h_C2[i]) > EPS) {
        printf("Mismatch at index %d, was: %f, should be: %f\n", i, h_C[i], h_C2[i]);
        return -1;
    }
  }
  printf("Success!\n");
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
  delete[] h_C2;

  return 0;
}
  
